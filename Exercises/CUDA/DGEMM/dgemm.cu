
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

#define epsilon (float) 1e-5
#define THREADxBLOCKalongXorY 16

typedef float DataType_t;

//
// Kernels
//
void MatrixMulOnHost(DataType_t* M, DataType_t* N, DataType_t* P, int Width)
{
    int i, j, k;
    DataType_t pvalue;

    for (i = 0; i < Width; i++)
    {
        for (j = 0; j < Width; j++)
        {
            pvalue = 0;
      
            for (k = 0; k < Width; k++)
                pvalue += M[i * Width + k] * N[k * Width + j];

            P[i*Width + j] = pvalue;
        }
    }
}

__global__ void MatrixMulKernel(DataType_t* dM, DataType_t* dN, DataType_t* dP, int Width)
{
    int i, j, k;
    DataType_t pvalue;

    i = blockIdx.y * blockDim.y + threadIdx.y;
    j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < Width && j < Width)
    {
        pvalue = 0;

        for (k = 0; k < Width; k++)
            pvalue += dM[i*Width + k] * dN[k*Width + j];
        
        dP[i*Width + j] = pvalue;
    }
}

void MatrixMulOnDevice(DataType_t* M, DataType_t* N, DataType_t* P, int Width)
{
    int gridsize, size;
    float mflops;
    DataType_t *dM, *dN, *dP;

    hipError_t mycudaerror;
    hipEvent_t start, stop;
    float elapsed;

    size = Width * Width * sizeof(DataType_t);

    // CUDA grid management
    gridsize = Width / THREADxBLOCKalongXorY;

    if (gridsize * THREADxBLOCKalongXorY < Width)
        gridsize = gridsize + 1;
    
    dim3 dimGrid(gridsize, gridsize);
    dim3 dimBlock(THREADxBLOCKalongXorY, THREADxBLOCKalongXorY);
    printf("Gridsize: %d\n", gridsize);

    hipMalloc(&dM, size);
    hipMemcpy(dM, M, size, hipMemcpyHostToDevice);
    hipMalloc(&dN, size);
    hipMemcpy(dN, N, size, hipMemcpyHostToDevice);
    hipMalloc(&dP, size);

    // cudaGetLastError call to reset previous CUDA errors
    mycudaerror = hipGetLastError();

    // Create start and stop CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Kernel launch
    hipEventRecord(start);
    MatrixMulKernel<<<dimGrid, dimBlock>>>(dM, dN, dP, Width);
    hipEventRecord(stop);

    // Device synchronization and cudaGetLastError call
    hipEventSynchronize(stop);

    // Event record, synchronization, elapsed time and destruction
    hipEventElapsedTime(&elapsed, start, stop);

    // calculate Mflops
    mflops = 2 * pow(Width, 3) / elapsed;
    elapsed /= 1000.f; // Convert to seconds
    
    printf("Kernel elapsed time %fs \n", elapsed);
    printf("Mflops: %f\n", mflops);

    // copy back results from device
    hipMemcpy(P, dP, size, hipMemcpyDeviceToHost);

    // free memory on device
    hipFree(dM);
    hipFree(dN);
    hipFree(dP);
}

//
// Main
//
int main(int argc, char** argv)
{
    int Width;
    DataType_t *M, *N, *hP, *gP;
    DataType_t it;
    int x, y;
    int errCnt;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s Width\n", argv[0]);
        exit(1);
    }

    Width = atoi(argv[1]);

    if (Width < 1)
    {
        fprintf(stderr, "Error Width=%d, must be > 0\n", Width);
        exit(1);
    }

    M = (DataType_t*) malloc(Width * Width * sizeof(DataType_t));
    N = (DataType_t*) malloc(Width * Width * sizeof(DataType_t));
    hP = (DataType_t*) malloc(Width * Width * sizeof(DataType_t));
    gP = (DataType_t*) malloc(Width * Width * sizeof(DataType_t));

    if (M == NULL)
    {
        fprintf(stderr,"Could not get memory for M\n");
        exit(1);
    }

    if (N == NULL)
    {
        fprintf(stderr,"Could not get memory for N\n");
        exit(1);
    }

    if (hP == NULL)
    {
        fprintf(stderr,"Could not get memory for hP\n");
        exit(1);
    }

    if (gP == NULL)
    {
        fprintf(stderr,"Could not get memory for gP\n");
        exit(1);
    }

    memset(gP, 0, Width * Width * sizeof(DataType_t));
    memset(hP, 0, Width * Width * sizeof(DataType_t));

    for (y = 0; y < Width; y++)
    {
        for (x = 0; x < Width; x++)
        {
            M[y*Width + x] = (DataType_t) (((y + 1) * Width + x + 1) / Width);
            N[y*Width + x] = (DataType_t) (((y + 1) * Width + x + 1) / Width);
        }
    }

    MatrixMulOnHost(M, N, hP, Width);
    MatrixMulOnDevice(M, N, gP, Width);

    errCnt = 0;

    for (y = 0; y < Width; y++)
    {
        for (x = 0; x < Width; x++)
        {
            it = hP[y*Width + x];
            
            if (fabs(it - gP[y*Width + x]) > epsilon*it)
            {
                printf("failing x=%d, y=%d: %f!=%f \n", x, y, it, gP[y*Width + x]);
                errCnt++;
            }
        }
    }

    if (errCnt == 0)
        printf("\nTEST PASSED\n");
    else
        printf("\n\nTEST FAILED: number of errors:  %d\n", errCnt);
}
