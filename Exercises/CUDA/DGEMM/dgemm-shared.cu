#include "hip/hip_runtime.h"
#include "sal.h"

#include <stdio.h>
#include <math.h>
#include <assert.h>

#define epsilon (float) 1e-5

#define BlockSize 32

typedef float DataType_t;

//
// Helpers
//
void MatrixRandomize(_Inout_ DataType_t* M, _In_ size_t Width)
{
    size_t i;

    for (i = 0; i < Width; i++)
        M[i] = (DataType_t) drand48();
}

// __attribute__((always inline))
// __device__

//
// Host and Device Kernels
//
__host__ void MatrixMulOnHost(DataType_t* M, DataType_t* N, DataType_t* P, int Width)
{
    int i, j, k;
    DataType_t pvalue;

    for (i = 0; i < Width; i++)
    {
        for (j = 0; j < Width; j++)
        {
            pvalue = 0;
      
            for (k = 0; k < Width; k++)
                pvalue += M[i * Width + k] * N[k * Width + j];

            P[i*Width + j] = pvalue;
        }
    }
}

__global__ void MatrixMulSharedKernel(
    _In_ DataType_t* dM,
    _In_ DataType_t* dN,
    _Out_ DataType_t* dP,
    _In_ size_t Width
)
/**
 * \brief Matrix-Matrix multiplication using shared mem
 *
 */
{
    __shared__ DataType_t As[BlockSize][BlockSize];
    __shared__ DataType_t Bs[BlockSize][BlockSize];

    DataType_t c;
    size_t it, jt, ib, jb;
    size_t k;

    it = threadIdx.y;
    jt = threadIdx.x;
    ib = blockIdx.y;
    jb = blockIdx.x;

    for (k = 0; k < Width / BlockSize; k++)
    {

    }
}

void MatrixMulOnDevice(DataType_t* M, DataType_t* N, DataType_t* P, size_t Width)
{
    hipEvent_t start, stop;
    DataType_t* d_A, *d_B, *d_C;
    size_t size;
    float gpu_time;
    double time_sec, num_ops, gflops;

    size = Width * Width * sizeof(float);

    // Load A and B to device memory 
    hipMalloc((void**) &d_A, size);
    hipMalloc((void**) &d_B, size);
 
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
 
    // Allocate C in device memory 
    hipMalloc((void**) &d_C, size);
 
    // Grid specify
    dim3 dimBlock(BlockSize, BlockSize); 
    dim3 dimGrid(Width / dimBlock.x, Width / dimBlock.x);
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start timing
    hipEventRecord(start);
 
    // Invoke kernel 
    MatrixMulSharedKernel <<<dimGrid, dimBlock>>> (d_A, d_B, d_C, Width);
 
    // End timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
 
    hipEventElapsedTime(&gpu_time, start, stop);
    time_sec = gpu_time / 1000.0;
    num_ops = 2.0 * (double) Width * (double) Width * (double) Width;
    gflops = 1.0e-9 * num_ops / time_sec;
    printf("CUDA Gflops = %.4f , Time = %.5f s dim=%d\n", gflops, time_sec, Width);
 
    // Read C from device memory 
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost); 
 
    // Free device memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

//
// Main
//
int main(int argc, char** argv)
{
    DataType_t* h_A, *h_B, *cpu_result, *gpu_result;
    size_t N, size;
    size_t i;
    int error;

    N = 32 * BlockSize;
    size = N * N * sizeof(DataType_t);

    // allocate matrices on the host
    h_A = (DataType_t*) malloc(size * sizeof(DataType_t));
    h_B = (DataType_t*) malloc(size * sizeof(DataType_t));

    // init matrices
    MatrixRandomize(h_A, N * N);
    MatrixRandomize(h_B, N * N);

    // allocate matrices to compare the results CPU/GPU
    cpu_result = (DataType_t*) malloc(size * sizeof(DataType_t));
    gpu_result = (DataType_t*) malloc(size * sizeof(DataType_t));

    // compute on GPU
    MatrixMulOnDevice(h_A, h_B, gpu_result, N);

    // compute on CPU
    MatrixMulOnHost(h_A, h_B, cpu_result, N);

    // check results
    error = 0;

    for (i = 0; i < N * N; i++)
    {
        if (fabs(cpu_result[i] - gpu_result[i]) > epsilon * cpu_result[i])
	        error++;
    }

    if (error == 0)
        printf("\nTEST PASSED\n");
    else
        printf("\n\nTEST FAILED: number of errors:  %d\n", error);

    free(h_A);
    free(h_B);
    free(cpu_result);
    free(gpu_result);
}
